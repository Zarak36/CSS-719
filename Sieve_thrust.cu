#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <iostream>
#include <chrono>

const int LIMIT = 100000000;  // Upper limit for the sieve
const int TARGET_PRIMES = 1000;  // Number of primes we want to find

// Custom functor to check if a number is prime
struct is_prime
{
    __host__ __device__
    bool operator()(const int x)
    {
        if (x < 2) return false;
        if (x == 2) return true;
        if (x % 2 == 0) return false;
       
        for (int i = 3; i * i <= x; i += 2)
        {
            if (x % i == 0) return false;
        }
        return true;
    }
};

// Functor to mark non-prime numbers in the sieve
struct mark_non_primes
{
    __host__ __device__
    void operator()(int& x)
    {
        if (x < 2)
        {
            x = 0;
            return;
        }
       
        for (int i = 2; i * i <= x; i++)
        {
            if (x % i == 0 && x != i)
            {
                x = 0;
                return;
            }
        }
    }
};

int main()
{
    auto start_time = std::chrono::high_resolution_clock::now();

    // Create a sequence of numbers from 0 to LIMIT-1
    thrust::device_vector<int> numbers(LIMIT);
    thrust::sequence(numbers.begin(), numbers.end());

    // Apply the sieve operation in parallel on GPU
    thrust::for_each(thrust::device, numbers.begin(), numbers.end(), mark_non_primes());

    // Remove all non-prime numbers (marked as 0)
    thrust::device_vector<int> primes(LIMIT);
    auto new_end = thrust::remove_copy(thrust::device,
                                     numbers.begin(), numbers.end(),
                                     primes.begin(),
                                     0);

    // Resize to actual number of primes found
    primes.resize(new_end - primes.begin());

    // Copy results back to host
    thrust::host_vector<int> host_primes = primes;

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_time = end_time - start_time;

    // Print the first 1000 primes
    int count = 0;
    for (size_t i = 0; i < std::min(size_t(TARGET_PRIMES), host_primes.size()); ++i) {
        std::cout << host_primes[i] << " ";
        if (++count % 10 == 0) std::cout << std::endl;
    }

    std::cout << "\nFound " << host_primes.size() << " prime numbers" << std::endl;
    std::cout << "Time taken: " << elapsed_time.count() << " seconds" << std::endl;

    return 0;
}