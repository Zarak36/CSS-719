#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>

const int LIMIT = 1000000;  // A large limit to ensure we find at least 1000 primes

__global__ void sieveKernel(bool* is_prime, int sqrt_limit) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 2;  // Start from 2
    if (idx <= sqrt_limit && is_prime[idx]) {
        for (int j = idx * idx; j < LIMIT; j += idx) {
            is_prime[j] = false;
        }
    }
}

int main() {
    // CPU start time
    auto cpu_start = std::chrono::high_resolution_clock::now();

    // Allocate memory on host
    bool* is_prime_h = new bool[LIMIT];
    std::fill_n(is_prime_h, LIMIT, true);
    is_prime_h[0] = is_prime_h[1] = false;  // 0 and 1 are not prime numbers

    // Allocate memory on device
    bool* is_prime_d;
    hipMalloc(&is_prime_d, LIMIT * sizeof(bool));
    hipMemcpy(is_prime_d, is_prime_h, LIMIT * sizeof(bool), hipMemcpyHostToDevice);

    // CUDA event to measure GPU time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    int sqrt_limit = static_cast<int>(std::sqrt(LIMIT));
    int threads_per_block = 256;
    int blocks = (LIMIT + threads_per_block - 1) / threads_per_block;
    sieveKernel<<<blocks, threads_per_block>>>(is_prime_d, sqrt_limit);
    hipDeviceSynchronize();

    // Stop GPU time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpu_milliseconds = 0;
    hipEventElapsedTime(&gpu_milliseconds, start, stop);

    // Copy results back to host
    hipMemcpy(is_prime_h, is_prime_d, LIMIT * sizeof(bool), hipMemcpyDeviceToHost);

    // CPU end time
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_duration = cpu_end - cpu_start;

    // Collect the first 1000 primes
    std::vector<int> primes;
    for (int i = 2; i < LIMIT && primes.size() < 1000; ++i) {
        if (is_prime_h[i]) {
            primes.push_back(i);
        }
    }

    // Output the first 1000 primes
    std::cout << "First 1000 primes:\n";
    for (size_t i = 0; i < primes.size(); ++i) {
        std::cout << primes[i] << " ";
        if ((i + 1) % 10 == 0) std::cout << "\n";
    }
    std::cout << "\n\n";

    // Output timing information
    std::cout << "Time taken on GPU: " << gpu_milliseconds / 1000.0 << " seconds\n";
    std::cout << "Time taken on CPU: " << cpu_duration.count() << " seconds\n";

    // Cleanup
    delete[] is_prime_h;
    hipFree(is_prime_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}